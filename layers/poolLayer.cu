#include"poolLayer.h"
#include"../config/config.h"
#include"../cuDNN_netWork.h"

#include"../tests/test_layer.h"
#include"opencv2/highgui.hpp"
#include"opencv2/core/core.hpp"
#include"opencv2/imgproc/imgproc.hpp"
using namespace cv;


void poolLayer:: createHandles()
	{
		/*hipdnnCreateTensorDescriptor创建一个tensor对象（并没有初始化）*/
		checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
		checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));
	}



poolLayer::poolLayer(string name)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	number = 0;
	channels = 0;
	height =0;
	width =0;
	lrate =  0.0f;
	prevLayer = NULL;
	nextLayer = NULL;

	configPooling* curConfig = (configPooling*) config::instanceObjtce()->getLayersByName(_name);
	string prevLayerName = curConfig->_input;
	convLayerBase* prev_Layer = (convLayerBase*) Layers::instanceObject()->getLayer(prevLayerName);

	nonLinearity = curConfig->_non_linearity;
	poolType = curConfig->_poolType;
	poolDim = curConfig->_size;
	pad_h = curConfig->_pad_h;
	pad_w = curConfig->_pad_w;
	stride_h =  curConfig->_stride_h;
	stride_w = curConfig->_stride_w;
    _inputImageDim = prev_Layer->_outputImageDim;
	/*池化后的大小*/
	_outputImageDim = _inputImageDim / poolDim;
	_inputAmount = prev_Layer->_outputAmount;
	_outputAmount = _inputAmount;
	outputSize = _outputAmount * _outputImageDim * _outputImageDim;

	this->createHandles();
}


 poolLayer::poolLayer(string name, const param_tuple& args)
{
	std::tie(poolType, poolDim, pad_h, pad_w, stride_h,
			stride_w, _inputImageDim, _inputAmount) = args;

	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	number = 0;
	channels = 0;
	height = 0;
	width = 0;
	lrate = 0.0f;
	prevLayer = NULL;
	nextLayer = NULL;

	_outputImageDim = _inputImageDim / poolDim;
	_outputAmount = _inputAmount;
	outputSize = _outputAmount * _outputImageDim * _outputImageDim;

	this->createHandles();
}


void poolLayer::forwardPropagation(string train_or_test)
{
	srcData = NULL;
	number = prevLayer->number;
	channels = prevLayer->channels;
	height = prevLayer->height;
	width = prevLayer->width;
	srcData = prevLayer->dstData;

	checkCUDNN(hipdnnSetPooling2dDescriptor(poolingDesc,
			                               HIPDNN_POOLING_MAX,
			                               poolDim,
			                               poolDim,//window
			                               pad_h,
			                               pad_w,//pading
			                               stride_h,
			                               stride_w));//stride


	/*根据池化设置相应的数据tensor*/
	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolingDesc,
    		                                     srcTensorDesc,
    		                                     &number,
    		                                     &channels,
    		                                     &height,
    		                                     &width));

	/*设置输出tensor*/

    checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
    		                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
    		                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
    		                              number,
    		                              channels,
    		                              height,
    		                              width));


	dstData = NULL;
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));

	float alpha = 1.0;
	float beta = 0.0;

	/*进行池化*/
	checkCUDNN(hipdnnPoolingForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                       poolingDesc,
			                       &alpha,
			                       srcTensorDesc,
			                       srcData,
			                       &beta,
			                       dstTensorDesc,
			                       dstData));


//	if(train_or_test == "test")
//		MemoryMonitor::instanceObject()->freeGpuMemory(srcData);
}

void poolLayer::Forward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(srcData);
}

void poolLayer::backwardPropagation(float Momentum)
{
   checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
		                                 cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                                 cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                                 number,
		                                 channels,
		                                 height,
		                                 width));
   checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
		                                cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                                cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                                number,
		                                channels,
		                                height,
		                                width));

   int prevlayer_n, prevlayer_c, prevlayer_h,prevlayer_w;
   prevlayer_n = prevLayer->number;
   prevlayer_c = prevLayer->channels;
   prevlayer_h = prevLayer->height;
   prevlayer_w = prevLayer->width;

   checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
		                               cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                               cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                               prevlayer_n,
		                               prevlayer_c,
		                               prevlayer_h,
		                               prevlayer_w));

   checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
		                                 cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                                 cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                                 prevlayer_n,
		                                 prevlayer_c,
		                                 prevlayer_h,
		                                 prevlayer_w));


   diffData = NULL;
   MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, prevlayer_n * prevlayer_c * prevlayer_h * prevlayer_w * sizeof(float));

   float alpha = 1.0f;
   float beta = 0.0;

   checkCUDNN(hipdnnPoolingBackward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
		                           poolingDesc,
		                           &alpha,
		                           dstTensorDesc,
		                           dstData,
		                           srcDiffTensorDesc,
		                           nextLayer->diffData,
		                           srcTensorDesc,
		                           srcData,
		                           &beta,
		                           dstDiffTensorDesc,
		                           diffData));


   //MemoryMonitor::instanceObject()->freeGpuMemory(dstData);
   //MemoryMonitor::instanceObject()->freeGpuMemory(nextLayer->diffData);
}



void poolLayer::Backward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(dstData);
	MemoryMonitor::instanceObject()->freeGpuMemory(nextLayer->diffData);
}

void poolLayer:: destroyHandles()
{
	/*销毁创建的描述符  逆向销毁*/
	checkCUDNN(hipdnnDestroyPoolingDescriptor(poolingDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc))
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));
}
