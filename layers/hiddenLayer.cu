#include "hip/hip_runtime.h"
#include"hiddenLayer.h"

void hiddenLayer::createHandles()
{
	hiprandCreateGenerator(&curandGenerator_W, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandCreateGenerator(&curandGenerator_B, HIPRAND_RNG_PSEUDO_MTGP32);
}

void hiddenLayer::destroyHandles()
{
	hiprandDestroyGenerator(curandGenerator_W);
	hiprandDestroyGenerator(curandGenerator_B);
}

void hiddenLayer::initRandom()
{
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, outputSize * inputSize * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, outputSize * 1 * 1 * 1 * sizeof(float));
	/*initial weight*/
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_W, time(NULL));
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_B, time(NULL));
	hiprandGenerateNormal(curandGenerator_W, dev_Weight, outputSize * inputSize, 0, epsilon);
	hiprandGenerateNormal(curandGenerator_B, dev_Bias, outputSize, 0, epsilon);

}

/*fill a float-point array with one*/
__global__ void FillOnes(float* vec, int value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > value) return ;

	vec[idx] = 1.0f;
}

/*constructor*/
hiddenLayer::hiddenLayer(string name, int sign)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	host_Weight = NULL;
	dev_Weight = NULL;
	host_Bias = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	VectorOnes = NULL;

    prevLayer.clear();
    nextLayer.clear();

	configHidden * curConfig = (configHidden*) config::instanceObjtce()->getLayersByName(_name);
	string preLayerName = curConfig->_input;
	layersBase* prev_Layer = (layersBase*) Layers::instanceObject()->getLayer(preLayerName);

	epsilon = curConfig->_init_w;
	lrate = curConfig->_lrate;
	inputSize = prev_Layer->getOutputSize();
	outputSize = curConfig->_NumHiddenNeurons;
	batchSize = config::instanceObjtce()->get_batchSize();
	lambda = curConfig->_weight_decay;

	inputAmount = prev_Layer->channels;
	inputImageDim = prev_Layer->height;
	prev_num = prev_Layer->number;
	prev_channels = prev_Layer->channels;
	prev_height = prev_Layer->height;
	prev_width = prev_Layer->width;
	number = prev_num;
	channels = outputSize;
	height = 1;
	width = 1;
	//1*batchSize
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&VectorOnes, 1 * 1 * 1 * batchSize* sizeof(float));
	FillOnes<<<1, batchSize>>>(VectorOnes, batchSize);

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad,1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad,1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, 1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad,1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad,1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, outputSize * batchSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData,  inputSize * batchSize* sizeof(float));

	this->createHandles();
	if(sign == RANDOM)
		this->initRandom();
}

void hiddenLayer::forwardPropagation(string train_or_test)
{
	srcData = prevLayer[0]->dstData;

	int dim_x = prev_channels * prev_height * prev_width ;
	int dim_y = outputSize ;
	float alpha = 1.0f;
	float beta = 0.0f;

	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_T,
				                  HIPBLAS_OP_N,
				                  dim_y,
				                  batchSize,
				                  dim_x,
				                  &alpha,
				                  dev_Weight,
				                  dim_x,
				                  srcData,
				                  dim_x,
				                  &beta,
				                  dstData,
				                  dim_y));

    //add bias
	alpha = 1.0f;
	beta = 1.0f;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_N,
				                  HIPBLAS_OP_N,
				                  dim_y,
				                  batchSize,
				                  1,
				                  &alpha,
				                  dev_Bias,
				                  dim_y,
				                  VectorOnes,
				                  1,
				                  &beta,
				                  dstData,
				                  dim_y));

	height = 1; width = 1; channels = dim_y;
}


void hiddenLayer::backwardPropagation(float Momentum)
{
	int dim_x = prev_channels * prev_height * prev_width;
	int dim_y = outputSize;

	checkCudaErrors(hipMemcpy(tmp_Wgrad, dev_Weight, 1 * 1 * outputSize * inputSize * sizeof(float), hipMemcpyDeviceToDevice));

	float alpha = 1.0f /(float)batchSize;
	float beta = lambda;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_N,
				                  HIPBLAS_OP_T,
				                  dim_x,
				                  dim_y,
				                  batchSize,
				                  &alpha,
				                  srcData,
				                  dim_x,
				                  nextLayer[0]->diffData,
				                  dim_y,
				                  &beta,
				                  tmp_Wgrad,
				                  dim_x));


	beta = 0.0f;
	checkCublasErrors(hipblasSgemv(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
			                      HIPBLAS_OP_N,
				                  outputSize,
				                  batchSize,
				                  &alpha,
				                  nextLayer[0]->diffData,
				                  outputSize,
				                  VectorOnes,
				                  1,
				                  &beta,
				                  tmp_Bgrad,
				                  1));

	alpha = 1.0f;
	beta = 0.0f;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
			                      HIPBLAS_OP_N,
			                      HIPBLAS_OP_N,
			                      dim_x,
				                  batchSize,
				                  outputSize,
				                  &alpha,
				                  dev_Weight,
				                  dim_x,
				                  nextLayer[0]->diffData,
				                  outputSize,
				                  &beta,
				                  diffData,
				                  dim_x));

	float scalVal = Momentum;
	int size = 1 * 1 * outputSize * inputSize * 1;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
							      size,
							      &scalVal,
							      dev_Wgrad,
							      1));


	size = 1 * 1 * outputSize * 1 * 1;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
								  size,
								  &scalVal,
								  dev_Bgrad,
								  1));

	scalVal = lrate;
	size = 1 * 1 * outputSize * inputSize * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
					                  size,
					                  &scalVal,
					                  tmp_Wgrad,
					                  1,
					                  dev_Wgrad,
					                  1));

	size = outputSize * 1 * 1 * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
						          size,
						          &scalVal,
						          tmp_Bgrad,
						          1,
						          dev_Bgrad,
						          1));

	/*updata weightt*/
	alpha = -1.0f;
	size = outputSize * inputSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  size,
				                  &alpha,
				                  dev_Wgrad,
				                  1,
				                  dev_Weight,
				                  1));

	size = outputSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  size,
				                  &alpha,
				                  dev_Bgrad,
				                  1,
				                  dev_Bias,
				                  1));
}


void hiddenLayer::saveWeight(FILE*file)
{
	host_Weight = NULL, host_Bias = NULL;
	copy_DeviceToHost(dev_Weight, host_Weight, 1, 1,outputSize, inputSize);
	copy_DeviceToHost(dev_Bias, host_Bias, 1, 1, 1, outputSize);

	for(int h = 0; h < outputSize; h++)
	{
		for(int w = 0; w < inputSize; w++)
		{
			fprintf(file, "%f ", host_Weight[w + inputSize * h]);
		}
	}

	for(int h = 0; h < outputSize; h++)
	{
		fprintf(file, "%f ", host_Bias[h]);
	}

	MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
	MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);

}


void hiddenLayer::readWeight(FILE*file)
{
	host_Weight = NULL; host_Bias = NULL;
	dev_Weight = NULL; dev_Bias = NULL;

	host_Weight = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(outputSize * inputSize * sizeof(float));
	host_Bias = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(outputSize * 1 * 1 * 1 * sizeof(float));

	for (int h = 0; h < outputSize; h++) {
		for (int w = 0; w < inputSize; w++) {
			fscanf(file, "%f", &host_Weight[w + inputSize * h]);
		}
	}

	for (int h = 0; h < outputSize; h++) {
		fscanf(file, "%f", &host_Bias[h]);
	}

	copy_HostToDevice(host_Weight, dev_Weight, 1, 1, outputSize, inputSize);
	copy_HostToDevice(host_Bias, dev_Bias, 1, 1, 1, outputSize);

	MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
	MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}
