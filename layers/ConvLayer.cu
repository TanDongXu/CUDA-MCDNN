#include "hip/hip_runtime.h"
#include"ConvLayer.h"
#include<hip/hip_runtime_api.h>
#include<glog/logging.h>

/*
 * Create handles
 * */
void ConvLayer::createHandles()
{
    checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    hiprandCreateGenerator(&curandGenerator_W, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandCreateGenerator(&curandGenerator_B, HIPRAND_RNG_PSEUDO_MTGP32);
}

/*
 * Destroy the handles
 * */
void ConvLayer:: destroyHandles()
{
    checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
    hiprandDestroyGenerator(curandGenerator_W);
   	hiprandDestroyGenerator(curandGenerator_B);
}

/*
 * Get the outputSize
 * */
int ConvLayer::getOutputSize()
{
   return outputSize;
}

/*
 * Random initial weights and Bias
 * */
void ConvLayer::initRandom()
{
    srand((unsigned)time(NULL));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));

    //set seed
    hiprandSetPseudoRandomGeneratorSeed(curandGenerator_W, time(NULL));
    hiprandSetPseudoRandomGeneratorSeed(curandGenerator_B, time(NULL));
    hiprandGenerateNormal(curandGenerator_W, dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize, 0, epsilon);
    //hiprandGenerateNormal(curandGenerator_B, dev_Bias, kernelAmount, 0, 0);

   // float* tmpWeight;
   // tmpWeight = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));

   // for(int n = 0; n < kernelAmount; n++)
   // {
   //     for(int c = 0; c < inputAmount; c++)
   //     {
   //         for(int h = 0; h < kernelSize; h++)
   //         {
   //             for(int w = 0; w < kernelSize; w++)
   //             {
   //                 tmpWeight[w + kernelSize * h + kernelSize * kernelSize * c + kernelSize * kernelSize * inputAmount * n] = epsilon * (2.0f * rand() / RAND_MAX - 1.0f);
   //             }
   //         }
   //     }
   // }

   // MemoryMonitor::instanceObject()->cpu2Gpu(dev_Weight, tmpWeight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));

   // delete tmpWeight;
}

/*
 * ConvLayer constructor
 * */
ConvLayer::ConvLayer(string name, int sign)
{
    _name = name;
    _inputName = " ";
    srcData = NULL;
    dstData = NULL;
    host_Weight = NULL;
    host_Bias = NULL;
    dev_Weight = NULL;
    dev_Bias = NULL;
    dev_Wgrad = NULL;
    dev_Bgrad = NULL;
    tmp_Wgrad = NULL;
    tmp_Bgrad = NULL;
    diffData = NULL;
    dev_weightSquare = NULL;
    host_weightSquare = NULL;
    prevLayer.clear();
    nextLayer.clear();

    filterDesc = NULL;
    convDesc = NULL;
    srcTensorDesc = NULL;
    dstTensorDesc = NULL;
    biasTensorDesc = NULL;
    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;

    /*can use class member prevLayer, because it has not assignment*/
    configConv* curConfig = (configConv*) config::instanceObjtce()->getLayersByName(_name);
    string prevLayerName = curConfig->_input;
    LayersBase* prev_Layer = (LayersBase*) Layers::instanceObject()->getLayer(prevLayerName);

    epsilon = curConfig->_init_w;
    lrate = curConfig->_lrate;
    batchSize = config::instanceObjtce()->get_batchSize();
    kernelAmount = curConfig->_kernelAmount;
    kernelSize = curConfig->_kernelSize;
    pad_h = curConfig->_pad_h;
    pad_w = curConfig->_pad_w;
    stride_h = curConfig->_stride_h;
    stride_w = curConfig->_stride_w;
    lambda = curConfig->_weight_decay;

    inputAmount = prev_Layer->channels;
    inputImageDim = prev_Layer->height;
    prev_num = prev_Layer->number;
    prev_channels = prev_Layer->channels;
    prev_height = prev_Layer->height;
    prev_width = prev_Layer->width;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    width = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    outputSize = channels * height * width;

    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_weightSquare = (float*) MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));

    this->createHandles();
    if(sign == RANDOM)
        this->initRandom();

    LOG(INFO) << "(" << number <<","<< channels << "," << height << "," << width <<")";
}

/*
 * Conv constructor overload
 * */
ConvLayer::ConvLayer(string name, int sign, const param_tuple& args)
{
    std::tie(pad_h, pad_w, stride_h, stride_w, kernelSize,
             kernelAmount, inputAmount, inputImageDim,
             epsilon, lrate, lambda) = args;

    _name = name;
    _inputName = " ";
    srcData = NULL;
    dstData = NULL;
    host_Weight = NULL;
    host_Bias = NULL;
    dev_Weight = NULL;
    dev_Bias = NULL;
    dev_Wgrad = NULL;
    dev_Bgrad = NULL;
    tmp_Wgrad = NULL;
    tmp_Bgrad = NULL;
    diffData = NULL;
    dev_weightSquare = NULL;
    host_weightSquare = NULL;
    prevLayer.clear();
    nextLayer.clear();

    filterDesc = NULL;
    convDesc = NULL;
    srcTensorDesc = NULL;
    dstTensorDesc = NULL;
    biasTensorDesc = NULL;
    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;

    batchSize = config::instanceObjtce()->get_batchSize();
    prev_num = config::instanceObjtce()->get_batchSize();
    prev_channels = inputAmount;
    prev_height = inputImageDim;
    prev_width = inputImageDim;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    width = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    outputSize = channels * height * width;

    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_weightSquare = (float*) MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));

    this->createHandles();
    if(sign == RANDOM)
        this->initRandom();
    LOG(INFO) << "(" << number <<","<< channels << "," << height << "," << width <<")";
}

/*
 * Deep copy constructor for convolution layers
 */
ConvLayer::ConvLayer(const ConvLayer* layer)
{
    srcData = NULL;
    dstData = NULL;
    host_Weight = NULL;
    host_Bias = NULL;
    dev_Weight = NULL;
    dev_Bias = NULL;
    dev_Wgrad = NULL;
    dev_Bgrad = NULL;
    tmp_Wgrad = NULL;
    tmp_Bgrad = NULL;
    diffData = NULL;
    dev_weightSquare = NULL;
    host_weightSquare = NULL;
    prevLayer.clear();
    nextLayer.clear();

    filterDesc = NULL;
    convDesc = NULL;
    srcTensorDesc = NULL;
    dstTensorDesc = NULL;
    biasTensorDesc = NULL;
    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;

    static int idx = 0;
    _name = layer->_name + string("_") + int_to_string(idx);
    idx ++;
    _inputName = layer->_inputName ;
    epsilon = layer->epsilon;
    lrate = layer->lrate;
    batchSize = layer->batchSize;
    kernelAmount = layer->kernelAmount;
    kernelSize = layer->kernelSize;
    pad_h = layer->pad_h;
    pad_w = layer->pad_w;
    stride_h = layer->stride_h;
    stride_w = layer->stride_w;
    lambda = layer->lambda;
    inputAmount = layer->inputAmount;
    inputImageDim = layer->inputImageDim;
    prev_num = layer->prev_num;
    prev_channels = layer->prev_channels;
    prev_height = layer->prev_height;
    prev_width = layer->prev_width;
    number = layer->number;
    channels = layer->channels;
    height = layer->height;
    width = layer->width;
    outputSize = layer->outputSize;

    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_weightSquare = (float*) MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    //    MemoryMonitor::instanceObject()->gpu2gpu(dev_Wgrad, layer->dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    //    MemoryMonitor::instanceObject()->gpu2gpu(dev_Bgrad, layer->dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    
    epsilon = 0.1;// use for cifar100
    this->createHandles();
    this->initRandom();

    //use for cifar100
    float* tWeight = NULL;
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tWeight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(tWeight, layer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    int size = kernelAmount * inputAmount * kernelSize * kernelSize;
    float alpha = 1.0f;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                 size,
                                 &alpha,
                                 tWeight,
                                 1,
                                 dev_Weight,
                                 1));
    
    LOG(INFO) << "(" << number <<","<< channels << "," << height << "," << width <<")";
    cout<<"Conv-copy"<<endl;
}


/*
 * Deep copy constructor for convolution layers
 */
ConvLayer::ConvLayer(const configBase* templateConfig)
{
    srcData = NULL;
    dstData = NULL;
    host_Weight = NULL;
    host_Bias = NULL;
    dev_Weight = NULL;
    dev_Bias = NULL;
    dev_Wgrad = NULL;
    dev_Bgrad = NULL;
    tmp_Wgrad = NULL;
    tmp_Bgrad = NULL;
    diffData = NULL;
    dev_weightSquare = NULL;
    host_weightSquare = NULL;
    prevLayer.clear();
    nextLayer.clear();

    filterDesc = NULL;
    convDesc = NULL;
    srcTensorDesc = NULL;
    dstTensorDesc = NULL;
    biasTensorDesc = NULL;
    convFwdAlgo = (hipdnnConvolutionFwdAlgo_t)-1;
    convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)-1;
    convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)-1;

    _name = templateConfig->_name;
    _inputName = templateConfig->_input;
    configConv* curConfig = (configConv*) templateConfig;
    LayersBase* prev_Layer = (LayersBase*) Layers::instanceObject()->getLayer(_inputName);

    epsilon = curConfig->_init_w;
    lrate = curConfig->_lrate;
    batchSize = config::instanceObjtce()->get_batchSize();
    kernelAmount = curConfig->_kernelAmount;
    kernelSize = curConfig->_kernelSize;
    pad_h = curConfig->_pad_h;
    pad_w = curConfig->_pad_w;
    stride_h = curConfig->_stride_h;
    stride_w = curConfig->_stride_w;
    lambda = curConfig->_weight_decay;

    inputAmount = prev_Layer->channels;
    inputImageDim = prev_Layer->height;
    prev_num = prev_Layer->number;
    prev_channels = prev_Layer->channels;
    prev_height = prev_Layer->height;
    prev_width = prev_Layer->width;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize) / stride_h + 1;
    width = (inputImageDim + 2 * pad_w - kernelSize) / stride_w + 1;
    outputSize = channels * height * width;
    
    // malloc GPU memory
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_weightSquare = (float*) MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    
    //find the same demension of conv layer weight from previous LayersBase
    configBase* findConfig = const_cast<configBase*>(templateConfig);
    ConvLayer* resultLayer = NULL;
    bool bFind = false;
    while(0 != findConfig->_prev.size())
    {
       if("CONV" == findConfig->_prev[0]->_type)
        {
            resultLayer = (ConvLayer*) Layers::instanceObject()->getLayer(findConfig->_prev[0]->_name);
            //must make sure kernelAmount kernelSize inputAmount equal respectively
            if((resultLayer->inputAmount == inputAmount) && (resultLayer->kernelSize == kernelSize) && (resultLayer->kernelAmount == kernelAmount))
            {
                bFind = true;
                break;
            }
        } 

        findConfig = findConfig->_prev[0];
    }

    this->createHandles();

    if(bFind)
    {
        CHECK(resultLayer);
        if(_name == "conv5" || _name == "conv7" || _name == "conv9" || _name == "conv11")
        {
            //if(_name == "conv3") epsilon = 0.01;
            if(_name == "conv5") epsilon = 0.01;
            if(_name == "conv7") epsilon = 0.005;
            if(_name == "conv9") epsilon = 0.005;
            if(_name == "conv11") epsilon = 0.005;
            this->initRandom();
            float* tWeight = NULL;
            MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tWeight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            MemoryMonitor::instanceObject()->gpu2gpu(tWeight, resultLayer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //MemoryMonitor::instanceObject()->gpu2gpu(dev_Weight, resultLayer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //reverseArray(dev_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
           // static int index = 0;
           // float scalVal = 3;
           // if(index == 1) scalVal = 1;
           // if(index == 2) scalVal = 0.5;
           // if(index == 3) scalVal = 0.1;
           int size =  kernelAmount * inputAmount * kernelSize * kernelSize;
           // checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
           //                         size,
           //                         &scalVal,
           //                         dev_Weight,
           //                         1));
           // MemoryMonitor::instanceObject()->gpu2gpu(dev_Bias, resultLayer->dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
            float alpha = 1.0f;
            checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                 size,
                                 &alpha,
                                 tWeight,
                                 1,
                                 dev_Weight,
                                 1));
        }else
        {
            MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
            MemoryMonitor::instanceObject()->gpu2gpu(dev_Weight, resultLayer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //float* tWeight = NULL;
            //MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tWeight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //MemoryMonitor::instanceObject()->gpu2gpu(tWeight, resultLayer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //MemoryMonitor::instanceObject()->gpu2gpu(dev_Weight, resultLayer->dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
            //reverseArray(dev_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
            //static int index = 0;
            //float scalVal = 3;
            //if(index == 1) scalVal = 1;
            //if(index == 2) scalVal = 0.5;
            //if(index == 3) scalVal = 0.1;
            //int size =  kernelAmount * inputAmount * kernelSize * kernelSize;
            //checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
            //                          size,
            //                          &scalVal,
            //                          dev_Weight,
            //                          1));
            //MemoryMonitor::instanceObject()->gpu2gpu(dev_Bias, resultLayer->dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
            //float alpha = 0.5f;
            // checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
            //                           size,
            //                           &alpha,
            //                           tWeight,
            //                           1,
            //                           dev_Weight,
            //                           1));
        }

        MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
        cout<<epsilon<<endl;
        cout<< _name<<endl;
        cout<<"copy weight"<<endl;
    }else
    {
        this->initRandom();
        cout<<"random weight"<<endl;
        //printf_DevParameter(kernelAmount,inputAmount, kernelSize, kernelSize, dev_Weight);
    }

    LOG(INFO) << "(" << number <<","<< channels << "," << height << "," << width <<")";
    cout<<"conv_copy"<<endl;
}

/*
 * Destructor
 * */
ConvLayer::~ConvLayer()
{
    MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
    MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
    MemoryMonitor::instanceObject()->freeGpuMemory(dev_Weight);
    MemoryMonitor::instanceObject()->freeGpuMemory(dev_Bias);
    MemoryMonitor::instanceObject()->freeGpuMemory(dev_Wgrad);
    MemoryMonitor::instanceObject()->freeGpuMemory(dev_Bgrad);
    MemoryMonitor::instanceObject()->freeGpuMemory(tmp_Wgrad);
    MemoryMonitor::instanceObject()->freeGpuMemory(tmp_Bgrad);
    MemoryMonitor::instanceObject()->freeGpuMemory(dstData);
    MemoryMonitor::instanceObject()->freeGpuMemory(diffData);
    destroyHandles();
}

// ReShape the demension int the Forward
void ConvLayer::ReShape()
{
    LayersBase* prev_Layer = (LayersBase*) Layers::instanceObject()->getLayer(_inputName);
    inputAmount = prev_Layer->channels;
    inputImageDim = prev_Layer->height;
    prev_channels = prev_Layer->channels;
    prev_height = prev_Layer->height;
    prev_width = prev_Layer->width;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize) / stride_h + 1;
    width = (inputImageDim + 2 * pad_w - kernelSize) / stride_w + 1;
    outputSize = channels * height * width;
}

//__global__ void compute_array_square(float* array, float* outArray, int size)
//{
//    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
//    int num_threads = blockDim.x * gridDim.x;
//    for(int i = 0; i < size; i += num_threads)
//    {
//        int index = i + thread_index;
//        if(index < size)
//        {
//            outArray[index] = array[index] * array[index];
//        }
//    }
//}

//compute cost
void ConvLayer::compute_cost()
{
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->cpuMemoryMemset(host_weightSquare, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    int size = kernelAmount * inputAmount * kernelSize * kernelSize;
    int threadsPerBlock = 256;
    int blockPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    compute_array_square<<<blockPerGrid, threadsPerBlock>>>(dev_Weight, dev_weightSquare, size);
    hipDeviceSynchronize();
    MemoryMonitor::instanceObject()->gpu2cpu(host_weightSquare, dev_weightSquare, size * sizeof(float));

    float tmpSum = 0.0f;
    for(int i = 0; i < size; i++)
    {
        tmpSum += host_weightSquare[i];
    }
    m_fCost = tmpSum * lambda / 2;
    //cout<<"conv: "<<m_fCost<<endl;
}

/*
 * Forward propagation add Bias
 */
void ConvLayer::addBias(const hipdnnTensorDescriptor_t& dstTensorDesc, int c, float *data )
{

    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDesc,
                                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
                                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
                                          1,
                                          c,
                                          1,
                                          1));

    float alpha = 1.0;
    float beta = 1.0;
    checkCUDNN(hipdnnAddTensor(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                              &alpha,
                              biasTensorDesc,
                              dev_Bias,
                              &beta,
                              dstTensorDesc,
                              data));
}

void ConvLayer::copyWeight()
{
    int size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    if(_name == "conv6")
    {
        ConvLayer* prev_Layer = (ConvLayer*) Layers::instanceObject()->getLayer("conv5");
        
        float alpha = 1.0f;
        checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  prev_Layer->dev_Weight,
                                  1,
                                  dev_Weight,
                                  1));

        cout<<"conv6 copy"<<endl;
    }
    if(_name == "conv5")
    {
        ConvLayer* prev_Layer = (ConvLayer*) Layers::instanceObject()->getLayer("conv4");
        
        float alpha = 1.0f;
        checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  prev_Layer->dev_Weight,
                                  1,
                                  dev_Weight,
                                  1));

        cout<<"conv5 copy"<<endl;
    }
    if(_name == "conv4")
    {
        ConvLayer* prev_Layer = (ConvLayer*) Layers::instanceObject()->getLayer("conv3");
        
        float alpha = 1.0f;
        checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  prev_Layer->dev_Weight,
                                  1,
                                  dev_Weight,
                                  1));

        cout<<"conv4 copy"<<endl;
    }
    if(_name == "conv3")
    {
        ConvLayer* prev_Layer = (ConvLayer*) Layers::instanceObject()->getLayer("conv2");
        
        float alpha = 1.0f;
        checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  prev_Layer->dev_Weight,
                                  1,
                                  dev_Weight,
                                  1));

        cout<<"conv3 copy"<<endl;
    }
}

/*
 * Convolution forward propagation
 * */
void ConvLayer::forwardPropagation(string train_or_test)
{
    srcData = prevLayer[0]->dstData;
    // dynamic adjust demension
    ReShape();
    //reverseArray(dev_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
                                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
                                          prev_num,
                                          prev_channels,
                                          prev_height,
                                          prev_width));

    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
                                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
                                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
                                          kernelAmount,
                                          inputAmount,
                                          kernelSize,
                                          kernelSize));

    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                               pad_h,
                                               pad_w,//pading
                                               stride_h,
                                               stride_w,//stride
                                               1,1,//upscale
                                               HIPDNN_CROSS_CORRELATION));

    checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
                                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
                                          number,
                                          channels,
                                          height,
                                          width));

    /*
     * Obtain the best suited algorithm for cudnnConvolutinForward
     * */
    //if (cuDNN_netWork<float>::instanceObject()->getConvFwdAlgorithm() < 0)
    //{
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                                       srcTensorDesc,
                                                       filterDesc,
                                                       convDesc,
                                                       dstTensorDesc,
                                                       HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                       0,
                                                       &convFwdAlgo));

        //cuDNN_netWork<float>::instanceObject()->setConvolutionFwdAlgorithm(convFwdAlgo);
    //}else
    //{
    //	convFwdAlgo =(hipdnnConvolutionFwdAlgo_t)cuDNN_netWork<float>::instanceObject()->getConvFwdAlgorithm();
    //}

    /*Get the amount of GPU memory for hipdnnConvolutionForward*/
    size_t convFwdSizeInBytes = 0;
    void* convFwdWorkSpace = NULL;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                                       srcTensorDesc,
                                                       filterDesc,
                                                       convDesc,
                                                       dstTensorDesc,
                                                       convFwdAlgo,
                                                       &convFwdSizeInBytes));

    if (convFwdSizeInBytes != 0)
    {
        checkCudaErrors(hipMalloc(&convFwdWorkSpace, convFwdSizeInBytes));
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    checkCUDNN(hipdnnConvolutionForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                       &alpha,
                                       srcTensorDesc,
                                       srcData,
                                       filterDesc,
                                       dev_Weight,
                                       convDesc,
                                       convFwdAlgo,
                                       convFwdWorkSpace,
                                       convFwdSizeInBytes,
                                       &beta,
                                       dstTensorDesc,
                                       dstData));

    /*add bias*/
    addBias(dstTensorDesc, channels, dstData);

    if (convFwdSizeInBytes != 0)
    {
        	checkCudaErrors(hipFree(convFwdWorkSpace));
    }
}

/*
 * Convolution backward propagation
 * */
void ConvLayer::backwardPropagation(float Momentum)
{
    /*Get the convolutuion function gradient with respect to the bias*/
    float alpha = 1.0f;
    float beta = 0.0f;
    int nIndex = m_nCurBranchIndex;
    checkCUDNN(hipdnnConvolutionBackwardBias(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                            &alpha,
                                            dstTensorDesc,
                                            nextLayer[nIndex]->diffData,
                                            &beta,
                                            biasTensorDesc,
                                            tmp_Bgrad
                                           ));

    /*Obtain the best suited algorithm for hipdnnConvolutionBackwardFilter*/
    //if(cuDNN_netWork<float>::instanceObject()->getConvolutionBwdFilterAlgorithm() < 0)
    //{
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
    			                                               srcTensorDesc,
    			                                               dstTensorDesc,
    			                                               convDesc,
    			                                               filterDesc,
    			                                               HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
    			                                               0,
    			                                               &convBwdFilterAlgo
    			                                               ));

    //	cuDNN_netWork<float>::instanceObject()->setConvolutionBwdFilterAlgorithm(convBwdFilterAlgo);
    //}else
    //{
    //	convBwdFilterAlgo = (hipdnnConvolutionBwdFilterAlgo_t)cuDNN_netWork<float>::instanceObject()->getConvolutionBwdFilterAlgorithm();
    //}

    /*Get the GPU memory workspace for hipdnnConvolutionBackwardFilter*/
    size_t convBwdFilterSizeInBytes = 0;
    void* convBwdFilterWorkSpace = NULL;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
    		                                                  srcTensorDesc,
    		                                                  dstTensorDesc,
    		                                                  convDesc,
    		                                                  filterDesc,
    		                                                  convBwdFilterAlgo,
    		                                                  &convBwdFilterSizeInBytes
    /*Alloc GPU memory*/		                                                  ));
    if(convBwdFilterSizeInBytes != 0)
    {
    	checkCudaErrors(hipMalloc(&convBwdFilterWorkSpace, convBwdFilterSizeInBytes));
    }

   /*This function computes the convolution gradient with respect to filter coefficient using the specified algo*/
    checkCUDNN(hipdnnConvolutionBackwardFilter(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                              &alpha,
                                              srcTensorDesc,
                                              srcData,
                                              dstTensorDesc,
                                              nextLayer[nIndex]->diffData,
                                              convDesc,
                                              convBwdFilterAlgo,
                                              convBwdFilterWorkSpace,
                                              convBwdFilterSizeInBytes,
                                              &beta,
                                              filterDesc,
                                              tmp_Wgrad));

    if (convBwdFilterSizeInBytes != 0)
    {
        checkCudaErrors(hipFree(convBwdFilterWorkSpace));
    }

    /*Obtaining the best suited algorithm for the hipdnnConvolutionBackwardData*/
    //if(cuDNN_netWork<float>::instanceObject()->getConvolutionBwdDataAlgorithm() < 0)
    //{
    checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
    			                                            filterDesc,
    			                                            dstTensorDesc,
    			                                            convDesc,
    			                                            dstTensorDesc,
    			                                            HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
    			                                            0,
    			                                            &convBwdDataAlgo
    			                                            ));
    //	cuDNN_netWork<float>::instanceObject()->setConvolutionBwdDataAlgorithm(convBwdDataAlgo);

    //}else
    //{
    //	convBwdDataAlgo = (hipdnnConvolutionBwdDataAlgo_t)cuDNN_netWork<float>::instanceObject()->getConvolutionBwdDataAlgorithm();
    //}

    /*Get the amount of GPU memory for the cudnnConvlotionBackwardData*/
    size_t convBwdDataSizeInBytes = 0;
    void* convBwdDataWorkSpace = NULL;
    /*按照接口说明srcTensorDesc应该是dstTensorDesc的,参考一个代码是用srcTensorDesc*/
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
    		                                                filterDesc,
    		                                                dstTensorDesc,
    		                                                convDesc,
    		                                                srcTensorDesc,
    		                                                convBwdDataAlgo,
    		                                                &convBwdDataSizeInBytes
    		                                                ));
    if(convBwdDataSizeInBytes != 0)
    {
    	checkCudaErrors(hipMalloc(&convBwdDataWorkSpace, convBwdDataSizeInBytes));
    }

    //Note:if use convBwdDataAlgo above,it will return error in running.
    // convBwdDataAlgo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
    /*Compute the convolution gradient with respect to the output tensor using the specified algo*/
    alpha = 1.0f;
    beta = 0.0f;
    checkCUDNN(hipdnnConvolutionBackwardData(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
                                            &alpha,
                                            filterDesc,
                                            dev_Weight,
                                            dstTensorDesc,
                                            nextLayer[nIndex]->diffData,
                                            convDesc,
                                            convBwdDataAlgo,
                                            convBwdDataWorkSpace,
                                            convBwdDataSizeInBytes,
                                            &beta,
                                            srcTensorDesc,
                                            diffData));

    if(convBwdDataSizeInBytes != 0)
    {
    	checkCudaErrors(hipFree(convBwdDataWorkSpace));
    }

    //compute cost
    compute_cost();

    /*
     * Update the weights in conv layer
     *
     * */
    alpha = lambda * batchSize;
    int size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  dev_Weight,
                                  1,
                                  tmp_Wgrad,
                                  1));

    float scalVal = Momentum;
    size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  dev_Wgrad,
                                  1));

    size = kernelAmount * 1 * 1 * 1;
    checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  dev_Bgrad,
                                  1));

    scalVal = lrate * 1.0f / batchSize;
    size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  tmp_Wgrad,
                                  1,
                                  dev_Wgrad,
                                  1));

    scalVal = 2 * lrate * 1.0f / batchSize;
    size = kernelAmount * 1 * 1 * 1;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &scalVal,
                                  tmp_Bgrad,
                                  1,
                                  dev_Bgrad,
                                  1));

    alpha = -1.0f;
    size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  dev_Wgrad,
                                  1,
                                  dev_Weight,
                                  1));

    size = kernelAmount * 1 * 1 * 1;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
                                  size,
                                  &alpha,
                                  dev_Bgrad,
                                  1,
                                  dev_Bias,
                                  1));
}

/*
 * Save the weights and Bias
 * */
void ConvLayer::saveWeight(FILE*file)
{
    host_Weight = NULL; host_Bias = NULL;
    copy_DeviceToHost(dev_Weight, host_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
    copy_DeviceToHost(dev_Bias, host_Bias, 1, 1, 1, kernelAmount);

    for(int n = 0 ; n < kernelAmount; n++)
    {
        for(int c = 0; c < inputAmount; c++)
        {
            for(int h = 0; h < kernelSize; h++)
            {
                for(int w = 0; w < kernelSize; w++)
                {
                    fprintf(file, "%f ", host_Weight[w + h * kernelSize + kernelSize * kernelSize * c + kernelSize * kernelSize * inputAmount * n]);
                }
            }
        }
    }

    for(int n = 0; n < kernelAmount; n++)
    {
        fprintf(file, "%f ", host_Bias[n]);
    }

    MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
    MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}

/*
 * Read the weights and Bias from file
 * */
void ConvLayer::readWeight(FILE*file)
{
    host_Weight = NULL; host_Bias = NULL;
    dev_Weight = NULL; dev_Bias = NULL;
    host_Weight = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_Bias = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * 1 * 1 * 1 * sizeof(float));

    for(int n = 0 ; n < kernelAmount; n++)
    {
        for(int c = 0; c < inputAmount; c++)
        {
            for(int h = 0; h < kernelSize; h++)
            {
                for(int w = 0; w < kernelSize; w++)
                {
                    fscanf(file, "%f", &host_Weight[w + h * kernelSize + kernelSize * kernelSize * c + kernelSize * kernelSize * inputAmount * n]);
                }
            }
        }
    }

    for (int n = 0; n < kernelAmount; n++)
    {
        fscanf(file, "%f", &host_Bias[n]);
    }

    copy_HostToDevice(host_Weight, dev_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
    copy_HostToDevice(host_Bias, dev_Bias, 1, 1, 1, kernelAmount);

    MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
    MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}
