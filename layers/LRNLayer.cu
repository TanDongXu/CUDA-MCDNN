#include"LRNLayer.h"
#include"../common/checkError.h"
#include"../config/config.h"
#include"../cuDNN_netWork.h"

void LRNLayer::createHandles()
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
	checkCUDNN(hipdnnCreateLRNDescriptor(&normDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));

}



LRNLayer::LRNLayer(string name)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	number = 0;
	channels = 0;
	height = 0;
	width = 0;
	lrate = 0.0f;
	prevLayer.clear();
	nextLayer.clear();

	configLRN* curConfig = (configLRN*)config::instanceObjtce()->getLayersByName(_name);
	string prevLayerName = curConfig->_input;
	layersBase* prev_Layer = (layersBase*)Layers::instanceObject()->getLayer(prevLayerName);

	lrnN = curConfig->_lrnN;
	lrnAlpha = curConfig->_lrnAlpha;
	lrnBeta = curConfig->_lrnBeta;
	lrnK = 1.0;


	inputSize = prev_Layer->getOutputSize();
	outputSize =inputSize;

	this->createHandles();
}



void LRNLayer::forwardPropagation(string train_or_test)
{
    srcData = NULL;
	number = prevLayer[0]->number;
	channels = prevLayer[0]->channels;
	height = prevLayer[0]->height;
	width = prevLayer[0]->width;
	srcData = prevLayer[0]->dstData;


	checkCUDNN(hipdnnSetLRNDescriptor(normDesc,
			                         lrnN,
			                         lrnAlpha,
			                         lrnBeta,
			                         lrnK));



	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	dstData = NULL;
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));


	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnLRNCrossChannelForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                               normDesc,
			                               HIPDNN_LRN_CROSS_CHANNEL,
			                               &alpha,
			                               srcTensorDesc,
			                               srcData,
			                               &beta,
			                               dstTensorDesc,
			                               dstData));

}


void LRNLayer::Forward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(srcData);
}


void LRNLayer::backwardPropagation(float Momentum)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));


	diffData = NULL;
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, number * channels * height* width * sizeof(float));

	float alpha = 1.0f;
	float beta = 0.0f;

	checkCUDNN(hipdnnLRNCrossChannelBackward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                                normDesc,
			                                HIPDNN_LRN_CROSS_CHANNEL,
			                                &alpha,
			                                dstTensorDesc,
			                                dstData,
			                                srcDiffTensorDesc,
			                                nextLayer[0]->diffData,
			                                srcTensorDesc,
			                                srcData,
			                                &beta,
			                                dstDiffTensorDesc,
			                                diffData));

}


void LRNLayer::Backward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(dstData);
	MemoryMonitor::instanceObject()->freeGpuMemory(nextLayer[0]->diffData);
}

void LRNLayer::destroyHandles()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc))
	checkCUDNN(hipdnnDestroyLRNDescriptor(normDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));

}
