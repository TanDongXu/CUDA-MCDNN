#include "hip/hip_runtime.h"
#include"hiddenLayer.h"

void hiddenLayer::createHandles()
{
	hiprandCreateGenerator(&curandGenerator_W, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandCreateGenerator(&curandGenerator_B, HIPRAND_RNG_PSEUDO_MTGP32);
}

void hiddenLayer::destroyHandles()
{
	hiprandDestroyGenerator(curandGenerator_W);
	hiprandDestroyGenerator(curandGenerator_B);
}

void hiddenLayer::initRandom()
{
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, outputSize * inputSize * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, outputSize * 1 * 1 * 1 * sizeof(float));
	/*initial weight*/
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_W, time(NULL));
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_B, time(NULL));
	hiprandGenerateNormal(curandGenerator_W, dev_Weight, outputSize * inputSize, 0, epsilon);
	hiprandGenerateNormal(curandGenerator_B, dev_Bias, outputSize, 0, epsilon);

}

/*fill a float-point array with one*/
__global__ void FillOnes(float* vec, int value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > value) return ;

	vec[idx] = 1.0f;
}

/*constructor*/
hiddenLayer::hiddenLayer(string name, int sign)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	host_Weight = NULL;
	dev_Weight = NULL;
	host_Bias = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	VectorOnes = NULL;

    prevLayer.clear();
    nextLayer.clear();

	configHidden * curConfig = (configHidden*) config::instanceObjtce()->getLayersByName(_name);
	string preLayerName = curConfig->_input;
	layersBase* prev_Layer = (layersBase*) Layers::instanceObject()->getLayer(preLayerName);

	epsilon = curConfig->_init_w;
	lrate = curConfig->_lrate;
	inputSize = prev_Layer->getOutputSize();
	outputSize = curConfig->_NumHiddenNeurons;
	batchSize = config::instanceObjtce()->get_batchSize();
	lambda = curConfig->_weight_decay;

	inputAmount = prev_Layer->channels;
	inputImageDim = prev_Layer->height;
	prev_num = prev_Layer->number;
	prev_channels = prev_Layer->channels;
	prev_height = prev_Layer->height;
	prev_width = prev_Layer->width;
	number = prev_num;
	channels = outputSize;
	height = 1;
	width = 1;

	printf("copy batchSize %d channels %d", batchSize, channels);
	printf("outputSize %d inputSize %d ", outputSize, inputSize);
	printf("prev_num %d prev_channels %d prev_height %d prev_width %d\n", prev_num, prev_channels, prev_height, prev_width);

	//1*batchSize
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&VectorOnes, 1 * 1 * 1 * batchSize* sizeof(float));
	FillOnes<<<1, batchSize>>>(VectorOnes, batchSize);
    hipDeviceSynchronize();

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Wgrad,1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Bgrad,1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, 1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Wgrad,1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Bgrad,1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dstData, outputSize * batchSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &diffData,  inputSize * batchSize* sizeof(float));

	this->createHandles();
	if(sign == RANDOM)
		this->initRandom();
}

//deep copy constructor
hiddenLayer::hiddenLayer(hiddenLayer* layer)
{
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	host_Weight = NULL;
	dev_Weight = NULL;
	host_Bias = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	VectorOnes = NULL;

	prevLayer.clear();
	nextLayer.clear();

	static int idx = 0;
	_name = layer->_name + int_to_string(idx);
	idx ++;
	_inputName = layer->_inputName;
	epsilon = layer->epsilon;
	lrate = layer->lrate;
	inputSize = layer->inputSize;
	outputSize = layer->outputSize;
	batchSize = layer->batchSize;
	lambda = layer->lambda;

	inputAmount = layer->inputAmount;
	inputImageDim = layer->inputImageDim;
	prev_num = layer->prev_num;
	prev_channels = layer->prev_channels;
	prev_height = layer->prev_height;
	prev_width = layer->prev_width;
	number = layer->number;
	channels = outputSize;
	height = 1;
	width = 1;

	//1*batchSize
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &VectorOnes, 1 * 1 * 1 * batchSize * sizeof(float));
	FillOnes<<<1, batchSize>>>(VectorOnes, batchSize);
	hipDeviceSynchronize();

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Weight, outputSize * inputSize * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Bias, outputSize * 1 * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Wgrad, 1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Bgrad, 1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Wgrad, 1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Bgrad, 1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dstData, outputSize * batchSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &diffData, inputSize * batchSize * sizeof(float));

	MemoryMonitor::instanceObject()->gpu2gpu(dev_Weight, layer->dev_Weight, outputSize * inputSize * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(dev_Bias, layer->dev_Bias, outputSize * 1 * 1 * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(dev_Wgrad, layer->dev_Wgrad, 1 * 1 * outputSize * inputSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(dev_Bgrad, layer->dev_Bgrad, 1 * 1 * outputSize * 1 * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(dstData, layer->dstData, outputSize * batchSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(diffData, layer->diffData, inputSize * batchSize * sizeof(float));

	this->createHandles();
	//cout<<"hidden copy"<<endl;
}



void hiddenLayer::forwardPropagation(string train_or_test)
{
	srcData = prevLayer[0]->dstData;

	int dim_x = prev_channels * prev_height * prev_width ;
	int dim_y = outputSize ;
	float alpha = 1.0f;
	float beta = 0.0f;

	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_T,
				                  HIPBLAS_OP_N,
				                  dim_y,
				                  batchSize,
				                  dim_x,
				                  &alpha,
				                  dev_Weight,
				                  dim_x,
				                  srcData,
				                  dim_x,
				                  &beta,
				                  dstData,
				                  dim_y));

    //add bias
	alpha = 1.0f;
	beta = 1.0f;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_N,
				                  HIPBLAS_OP_N,
				                  dim_y,
				                  batchSize,
				                  1,
				                  &alpha,
				                  dev_Bias,
				                  dim_y,
				                  VectorOnes,
				                  1,
				                  &beta,
				                  dstData,
				                  dim_y));

	height = 1; width = 1; channels = dim_y;

}


void hiddenLayer::backwardPropagation(float Momentum)
{
	int dim_x = prev_channels * prev_height * prev_width;
	int dim_y = outputSize;

	checkCudaErrors(hipMemcpy(tmp_Wgrad, dev_Weight, 1 * 1 * outputSize * inputSize * sizeof(float), hipMemcpyDeviceToDevice));

	float alpha = 1.0f /(float)batchSize;
	float beta = lambda;
	int nIndex = m_nCurBranchIndex;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  HIPBLAS_OP_N,
				                  HIPBLAS_OP_T,
				                  dim_x,
				                  dim_y,
				                  batchSize,
				                  &alpha,
				                  srcData,
				                  dim_x,
				                  nextLayer[nIndex]->diffData,
				                  dim_y,
				                  &beta,
				                  tmp_Wgrad,
				                  dim_x));


	beta = 0.0f;
	checkCublasErrors(hipblasSgemv(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
			                      HIPBLAS_OP_N,
				                  outputSize,
				                  batchSize,
				                  &alpha,
				                  nextLayer[nIndex]->diffData,
				                  outputSize,
				                  VectorOnes,
				                  1,
				                  &beta,
				                  tmp_Bgrad,
				                  1));

	alpha = 1.0f;
	beta = 0.0f;
	checkCublasErrors(hipblasSgemm(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
			                      HIPBLAS_OP_N,
			                      HIPBLAS_OP_N,
			                      dim_x,
				                  batchSize,
				                  outputSize,
				                  &alpha,
				                  dev_Weight,
				                  dim_x,
				                  nextLayer[nIndex]->diffData,
				                  outputSize,
				                  &beta,
				                  diffData,
				                  dim_x));

	float scalVal = Momentum;
	int size = 1 * 1 * outputSize * inputSize * 1;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
							      size,
							      &scalVal,
							      dev_Wgrad,
							      1));


	size = 1 * 1 * outputSize * 1 * 1;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
								  size,
								  &scalVal,
								  dev_Bgrad,
								  1));

	scalVal = lrate;
	size = 1 * 1 * outputSize * inputSize * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
					                  size,
					                  &scalVal,
					                  tmp_Wgrad,
					                  1,
					                  dev_Wgrad,
					                  1));

	size = outputSize * 1 * 1 * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
						          size,
						          &scalVal,
						          tmp_Bgrad,
						          1,
						          dev_Bgrad,
						          1));

	/*updata weightt*/
	alpha = -1.0f;
	size = outputSize * inputSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  size,
				                  &alpha,
				                  dev_Wgrad,
				                  1,
				                  dev_Weight,
				                  1));

	size = outputSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  size,
				                  &alpha,
				                  dev_Bgrad,
				                  1,
				                  dev_Bias,
				                  1));
}


void hiddenLayer::saveWeight(FILE*file)
{
	host_Weight = NULL, host_Bias = NULL;
	copy_DeviceToHost(dev_Weight, host_Weight, 1, 1,outputSize, inputSize);
	copy_DeviceToHost(dev_Bias, host_Bias, 1, 1, 1, outputSize);

	for(int h = 0; h < outputSize; h++)
	{
		for(int w = 0; w < inputSize; w++)
		{
			fprintf(file, "%f ", host_Weight[w + inputSize * h]);
		}
	}

	for(int h = 0; h < outputSize; h++)
	{
		fprintf(file, "%f ", host_Bias[h]);
	}

	MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
	MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);

}


void hiddenLayer::readWeight(FILE*file)
{
	host_Weight = NULL; host_Bias = NULL;
	dev_Weight = NULL; dev_Bias = NULL;

	host_Weight = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(outputSize * inputSize * sizeof(float));
	host_Bias = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(outputSize * 1 * 1 * 1 * sizeof(float));

	for (int h = 0; h < outputSize; h++) {
		for (int w = 0; w < inputSize; w++) {
			fscanf(file, "%f", &host_Weight[w + inputSize * h]);
		}
	}

	for (int h = 0; h < outputSize; h++) {
		fscanf(file, "%f", &host_Bias[h]);
	}

	copy_HostToDevice(host_Weight, dev_Weight, 1, 1, outputSize, inputSize);
	copy_HostToDevice(host_Bias, dev_Bias, 1, 1, 1, outputSize);

	MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
	MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}
