#include"convLayer.h"
#include<hip/hip_runtime_api.h>

void convLayer::createHandles()
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));

	hiprandCreateGenerator(&curandGenerator_W, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandCreateGenerator(&curandGenerator_B, HIPRAND_RNG_PSEUDO_MTGP32);
}

void convLayer::initRandom()
{
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, 
                                                      kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));

	//set seed
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_W, time(NULL));
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator_B, time(NULL));
	hiprandGenerateNormal(curandGenerator_W, dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize, 0, epsilon);
	hiprandGenerateNormal(curandGenerator_B, dev_Bias, kernelAmount, 0, epsilon);
}


/*convLayer constructor*/
convLayer::convLayer(string name, int sign)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	host_Weight = NULL;
	host_Bias = NULL;
	dev_Weight = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	diffData = NULL;
    prevLayer.clear();
    nextLayer.clear();

    /*can use class member prevLayer, because it has not assignment*/
	configConv* curConfig = (configConv*) config::instanceObjtce()->getLayersByName(_name);
    string prevLayerName = curConfig->_input;
    layersBase* prev_Layer = (layersBase*) Layers::instanceObject()->getLayer(prevLayerName);

    epsilon = curConfig->_init_w;
    lrate = curConfig->_lrate;
    batchSize = config::instanceObjtce()->get_batchSize();
    kernelAmount = curConfig->_kernelAmount;
    kernelSize = curConfig->_kernelSize;
    pad_h = curConfig->_pad_h;
    pad_w = curConfig->_pad_w;
    stride_h = curConfig->_stride_h;
    stride_w = curConfig->_stride_w;
    lambda = curConfig->_weight_decay;

    inputAmount = prev_Layer->channels;
    inputImageDim = prev_Layer->height;
    prev_num = prev_Layer->number;
    prev_channels = prev_Layer->channels;
    prev_height = prev_Layer->height;
    prev_width = prev_Layer->width;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    width = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    outputSize = channels * height * width;

    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad, 
                                                     kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, 
                                                     kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad,
                                                      kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData,
        		                                      batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData,
                                                      batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    this->createHandles();
    if(sign == RANDOM)
    	this->initRandom();
}


/*conv constructor overload*/
convLayer::convLayer(string name, int sign, const param_tuple& args)
{
	std::tie(pad_h, pad_w, stride_h, stride_w, kernelSize,
			kernelAmount, inputAmount, inputImageDim,
			epsilon, lrate, lambda) = args;

	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	host_Weight = NULL;
	host_Bias = NULL;
	dev_Weight = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	diffData = NULL;
    prevLayer.clear();
    nextLayer.clear();

    batchSize = config::instanceObjtce()->get_batchSize();
    prev_num = config::instanceObjtce()->get_batchSize();
    prev_channels = inputAmount;
    prev_height = inputImageDim;
    prev_width = inputImageDim;
    number = prev_num;
    channels = kernelAmount;
    height = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    width = (inputImageDim + 2 * pad_h - kernelSize)/stride_h + 1;
    outputSize = channels * height * width;

    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMemoryMemset(dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**) &diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));

	this->createHandles();
	if(sign == RANDOM)
		this->initRandom();
}

//deep copy constructor
convLayer::convLayer(convLayer* layer)
{
	srcData = NULL;
	dstData = NULL;
	host_Weight = NULL;
	host_Bias = NULL;
	dev_Weight = NULL;
	dev_Bias = NULL;
	dev_Wgrad = NULL;
	dev_Bgrad = NULL;
	tmp_Wgrad = NULL;
	tmp_Bgrad = NULL;
	diffData = NULL;
	prevLayer.clear();
	nextLayer.clear();

	static int idx = 0;
	_name = layer->_name + string("_") + int_to_string(idx);
	idx ++;
	_inputName = layer->_inputName ;
	epsilon = layer->epsilon;
	lrate = layer->lrate;
	batchSize = layer->batchSize;
	kernelAmount = layer->kernelAmount;
	kernelSize = layer->kernelSize;
	pad_h = layer->pad_h;
	pad_w = layer->pad_w;
	stride_h = layer->stride_h;
	stride_w = layer->stride_w;
	lambda = layer->lambda;
	inputAmount = layer->inputAmount;
	inputImageDim = layer->inputImageDim;
	prev_num = layer->prev_num;
	prev_channels = layer->prev_channels;
	prev_height = layer->prev_height;
	prev_width = layer->prev_width;
	number = layer->number;
	channels = layer->channels;
	height = layer->height;
	width = layer->width;
	outputSize = layer->outputSize;

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Weight, kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&tmp_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(dev_Weight,layer->dev_Weight,  kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(dev_Bias, layer->dev_Bias, kernelAmount * 1 * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(dev_Wgrad, layer->dev_Wgrad, kernelAmount * inputAmount * 1 * kernelSize * kernelSize * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(dev_Bgrad, layer->dev_Bgrad, 1 * kernelAmount * 1 * 1 * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(dstData, layer->dstData, batchSize * kernelAmount * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpu2gpu(diffData, layer->diffData, batchSize * inputAmount * inputImageDim * inputImageDim * sizeof(float));

    //cout<<"conv deep copy"<<endl;
    this->createHandles();
}

void convLayer::addBias(const hipdnnTensorDescriptor_t& dstTensorDesc, int c, float *data )
{
    
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          1,
                                          c,
				                          1,
				                          1));


	float alpha = 1.0;
	float beta = 1.0;
	checkCUDNN(hipdnnAddTensor(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                  CUDNN_ADD_SAME_C,
			                  &alpha,
			                  biasTensorDesc,
			                  dev_Bias,
			                  &beta,
			                  dstTensorDesc,
			                  data));
}


void convLayer::forwardPropagation(string train_or_test)
{
	srcData = prevLayer[0]->dstData;

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              prev_num,
			                              prev_channels,
			                              prev_height,
			                              prev_width));

	checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              kernelAmount,
			                              inputAmount,
			                              kernelSize,
			                              kernelSize));

	checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
			                                   pad_h,
			                                   pad_w,//pading
			                                   stride_h,
			                                   stride_w,//stride
			                                   1,1,//upscale
			                                   HIPDNN_CROSS_CORRELATION));


	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	if (cuDNN_netWork<float>::instanceObject()->GetconvAlgorithm() < 0)
	{

		checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
						                               srcTensorDesc,
						                               filterDesc,
						                               convDesc,
						                               dstTensorDesc,
						                               HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
						                               0,
						                               &algo));

	    cuDNN_netWork<float>::instanceObject()->setConvolutionAlgorithm(algo);

	}else
	{
		algo =(hipdnnConvolutionFwdAlgo_t)cuDNN_netWork<float>::instanceObject()->GetconvAlgorithm();
	}

	size_t sizeInBytes = 0;
	void* workSpace =NULL;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
					                                   srcTensorDesc,
					                                   filterDesc,
					                                   convDesc,
					                                   dstTensorDesc,
					                                   algo,
					                                   &sizeInBytes));

	if (sizeInBytes != 0)
	{
		checkCudaErrors(hipMalloc(&workSpace, sizeInBytes));
	}

	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnConvolutionForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
					                   &alpha,
					                   srcTensorDesc,
					                   srcData,
					                   filterDesc,
					                   dev_Weight,
					                   convDesc,
					                   algo,
					                   workSpace,
					                   sizeInBytes,
					                   &beta,
					                   dstTensorDesc,
					                   dstData));

	/*add bias*/
	addBias(dstTensorDesc, channels, dstData);

	if (sizeInBytes != 0)
	{
		checkCudaErrors(hipFree(workSpace));
	}
}


void convLayer::backwardPropagation(float Momentum)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              prev_num,
			                              prev_channels,
			                              prev_height,
			                              prev_width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              prev_num,
			                              prev_channels,
			                              prev_height,
			                              prev_width));

	float alpha = 1.0f;
	float beta = 0.0f;
	int nIndex = m_nCurBranchIndex;
	checkCUDNN(hipdnnConvolutionBackwardBias(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                                &alpha,
			                                srcDiffTensorDesc,
			                                nextLayer[nIndex]->diffData,
			                                &beta,
			                                biasTensorDesc,
			                                tmp_Bgrad
			                                ));

	checkCUDNN(hipdnnConvolutionBackwardFilter(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                                  &alpha,
			                                  srcTensorDesc,
			                                  srcData,
			                                  srcDiffTensorDesc,
			                                  nextLayer[nIndex]->diffData,
			                                  convDesc,
			                                  &beta,
			                                  filterDesc,
			                                  tmp_Wgrad));

	alpha = lambda * batchSize;
	int size =  kernelAmount * inputAmount * kernelSize * kernelSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
						          size,
						          &alpha,
						          dev_Weight,
						          1,
						          tmp_Wgrad,
						          1));
	alpha = 1.0f;
	beta = 0.0f;
	checkCUDNN(hipdnnConvolutionBackwardData(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                                &alpha,
			                                filterDesc,
			                                dev_Weight,
			                                srcDiffTensorDesc,
			                                nextLayer[nIndex]->diffData,
			                                convDesc,
			                                &beta,
			                                dstDiffTensorDesc,
			                                diffData));

	float scalVal = Momentum;
	size =  kernelAmount * inputAmount * kernelSize * kernelSize;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
						          size,
						          &scalVal,
						          dev_Wgrad,
						          1));

	size = kernelAmount * 1 * 1 * 1;
	checkCublasErrors(hipblasSscal(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
							      size,
							      &scalVal,
							      dev_Bgrad,
							      1));

	scalVal =lrate * 1.0f / batchSize;
	size =  kernelAmount * inputAmount * kernelSize * kernelSize;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
				                  size,
				                  &scalVal,
				                  tmp_Wgrad,
				                  1,
				                  dev_Wgrad,
				                  1));

	size = kernelAmount * 1 * 1 * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
					              size,
					              &scalVal,
					              tmp_Bgrad,
					              1,
					              dev_Bgrad,
					              1));
	
    alpha = -1.0f;
	size =  kernelAmount * inputAmount * kernelSize * kernelSize;
    checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
					                  size,
					                  &alpha,
					                  dev_Wgrad,
					                  1,
					                  dev_Weight,
					                  1));

	size = kernelAmount * 1 * 1 * 1;
	checkCublasErrors(hipblasSaxpy(cuDNN_netWork<float>::instanceObject()->GetcublasHandle(),
					                  size,
					                  &alpha,
					                  dev_Bgrad,
					                  1,
					                  dev_Bias,
					                  1));
}


void convLayer::saveWeight(FILE*file)
{
	host_Weight = NULL; host_Bias = NULL;
	copy_DeviceToHost(dev_Weight, host_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
	copy_DeviceToHost(dev_Bias, host_Bias, 1, 1, 1, kernelAmount);

	for(int n = 0 ; n < kernelAmount; n++)
	{
		for(int c = 0; c < inputAmount; c++)
		{
			for(int h = 0; h < kernelSize; h++)
			{
				for(int w = 0; w < kernelSize; w++)
				{
				   fprintf(file, "%f ", host_Weight[w + h * kernelSize + kernelSize * kernelSize * c + kernelSize * kernelSize * inputAmount * n]);
				}
			}
		}
	}

	for(int n = 0; n < kernelAmount; n++)
	{
		fprintf(file, "%f ", host_Bias[n]);
	}

	MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
	MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}



void convLayer::readWeight(FILE*file)
{
	host_Weight = NULL; host_Bias = NULL;
	dev_Weight = NULL; dev_Bias = NULL;
	host_Weight = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * inputAmount * kernelSize * kernelSize * sizeof(float));
    host_Bias = (float*)MemoryMonitor::instanceObject()->cpuMallocMemory(kernelAmount * 1 * 1 * 1 * sizeof(float));

    for(int n = 0 ; n < kernelAmount; n++)
    	{
    		for(int c = 0; c < inputAmount; c++)
    		{
    			for(int h = 0; h < kernelSize; h++)
    			{
    				for(int w = 0; w < kernelSize; w++)
    				{
    				   fscanf(file, "%f", &host_Weight[w + h * kernelSize + kernelSize * kernelSize * c + kernelSize * kernelSize * inputAmount * n]);
    				}
    			}
    		}
    	}

	for (int n = 0; n < kernelAmount; n++)
	{
		fscanf(file, "%f", &host_Bias[n]);
	}


	copy_HostToDevice(host_Weight, dev_Weight, kernelAmount, inputAmount, kernelSize, kernelSize);
    copy_HostToDevice(host_Bias, dev_Bias, 1, 1, 1, kernelAmount);

    MemoryMonitor::instanceObject()->freeCpuMemory(host_Weight);
    MemoryMonitor::instanceObject()->freeCpuMemory(host_Bias);
}


void convLayer:: destroyHandles()
{
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));
	hiprandDestroyGenerator(curandGenerator_W);
	hiprandDestroyGenerator(curandGenerator_B);
}






