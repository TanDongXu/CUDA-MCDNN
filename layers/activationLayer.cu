#include"activationLayer.h"
#include"../config/config.h"
#include"../cuDNN_netWork.h"
#include"../tests/test_layer.h"

#include"opencv2/imgproc/imgproc.hpp"
#include"opencv2/highgui/highgui.hpp"

using namespace cv;

void activationLayer::createHandles()
	{
		checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
		checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));
	}

/*activation layer constructor*/
activationLayer::activationLayer(string name)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	number = 0;
	channels =0;
	height = 0;
	width = 0;
	lrate = 0.0f;
	prevLayer = NULL;
	nextLayer = NULL;

	configActivation * curConfig = (configActivation*) config::instanceObjtce()->getLayersByName(_name);
	string preLayerName = curConfig->_input;
	//layersBase* prelayer = (layersBase*) Layers::instanceObject()->getLayer(preLayerName);

	convLayerBase* prev_Layer = (convLayerBase*) Layers::instanceObject()->getLayer(preLayerName);

	_inputAmount = prev_Layer->_outputAmount;
    _outputAmount = _inputAmount;
	_inputImageDim = prev_Layer->_outputImageDim;
	_outputImageDim = _inputImageDim;

	inputSize = prev_Layer->getOutputSize();
    outputSize =inputSize;

    this->createHandles();
}


void activationLayer::forwardPropagation(string train_or_test)
{
	number = prevLayer->number;
	channels = prevLayer->channels;
	height = prevLayer->height;
	width = prevLayer->width;
	srcData = prevLayer->dstData;

	dstData = NULL;
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));


    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
		                                 cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                                 cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                                 number,
		                                 channels,
		                                 height,
		                                 width));


	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnActivationForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                          HIPDNN_ACTIVATION_RELU,
			                          &alpha,
			                          srcTensorDesc,
			                          srcData,
			                          &beta,
			                          dstTensorDesc,
			                          dstData));

}


/*free forwardPropagation memory*/
void activationLayer::Forward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(srcData);
}


void activationLayer::backwardPropagation(float Momentum)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));


	checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));


	checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));


	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));





	diffData = NULL;
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, number * channels * height * width * sizeof(float));

	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnActivationBackward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                           HIPDNN_ACTIVATION_RELU,
			                           &alpha,
			                           dstTensorDesc,
			                           dstData,
			                           srcDiffTensorDesc,
			                           nextLayer->diffData,
			                           srcTensorDesc,
			                           srcData,
			                           &beta,
			                           dstDiffTensorDesc,
			                           diffData));

}


/*free backwardPropagation memory*/
void activationLayer::Backward_cudaFree()
{
	MemoryMonitor::instanceObject()->freeGpuMemory(dstData);
	MemoryMonitor::instanceObject()->freeGpuMemory(nextLayer->diffData);
}


void activationLayer::destroyHandles()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));

}
