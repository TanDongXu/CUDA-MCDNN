#include"LRNLayer.h"
#include"../common/checkError.h"
#include"../config/config.h"
#include"../cuDNN_netWork.h"

void LRNLayer::createHandles()
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
	checkCUDNN(hipdnnCreateLRNDescriptor(&normDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));
}

LRNLayer::LRNLayer(string name)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	lrate = 0.0f;
	prevLayer.clear();
	nextLayer.clear();

	configLRN* curConfig = (configLRN*)config::instanceObjtce()->getLayersByName(_name);
	string prevLayerName = curConfig->_input;
	layersBase* prev_Layer = (layersBase*)Layers::instanceObject()->getLayer(prevLayerName);

	lrnN = curConfig->_lrnN;
	lrnAlpha = curConfig->_lrnAlpha;
	lrnBeta = curConfig->_lrnBeta;
	lrnK = 1.0;

	inputAmount = prev_Layer->channels;
	inputImageDim = prev_Layer->height;
	number = prev_Layer->number;
	channels = prev_Layer->channels;
	height = prev_Layer->height;
	width = prev_Layer->width;
	inputSize = prev_Layer->getOutputSize();
	outputSize =inputSize;

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, number * channels * height* width * sizeof(float));

	this->createHandles();
}

void LRNLayer::forwardPropagation(string train_or_test)
{
	srcData = prevLayer[0]->dstData;

	checkCUDNN(hipdnnSetLRNDescriptor(normDesc,
			                         lrnN,
			                         lrnAlpha,
			                         lrnBeta,
			                         lrnK));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnLRNCrossChannelForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                               normDesc,
			                               HIPDNN_LRN_CROSS_CHANNEL,
			                               &alpha,
			                               srcTensorDesc,
			                               srcData,
			                               &beta,
			                               dstTensorDesc,
			                               dstData));

}


void LRNLayer::backwardPropagation(float Momentum)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnLRNCrossChannelBackward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                                normDesc,
			                                HIPDNN_LRN_CROSS_CHANNEL,
			                                &alpha,
			                                dstTensorDesc,
			                                dstData,
			                                srcDiffTensorDesc,
			                                nextLayer[0]->diffData,
			                                srcTensorDesc,
			                                srcData,
			                                &beta,
			                                dstDiffTensorDesc,
			                                diffData));

}


void LRNLayer::destroyHandles()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc))
	checkCUDNN(hipdnnDestroyLRNDescriptor(normDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));
}
