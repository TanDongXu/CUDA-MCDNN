#include"activationLayer.h"
#include"../config/config.h"
#include"../cuDNN_netWork.h"
#include"../tests/test_layer.h"
#include"../common/utility.cuh"

void activationLayer::createHandles()
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcDiffTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstDiffTensorDesc));
}

/*activation layer constructor*/
activationLayer::activationLayer(string name)
{
	_name = name;
	_inputName = " ";
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
    prevLayer.clear();
    nextLayer.clear();

	configActivation * curConfig = (configActivation*) config::instanceObjtce()->getLayersByName(_name);
	string preLayerName = curConfig->_input;
	layersBase* prev_Layer = (layersBase*) Layers::instanceObject()->getLayer(preLayerName);

	inputAmount = prev_Layer->channels;
	inputImageDim = prev_Layer->height;
	number = prev_Layer->number;
	channels = prev_Layer->channels;
	height = prev_Layer->height;
	width = prev_Layer->width;
    outputSize = channels * height * width;

    ActivationMode = (hipdnnActivationMode_t)curConfig->_non_linearity;
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));
    MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, number * channels * height * width * sizeof(float));

    this->createHandles();
}

//deep copy constructor
activationLayer::activationLayer(activationLayer* layer)
{
	srcData = NULL;
	dstData = NULL;
	diffData = NULL;
	prevLayer.clear();
	nextLayer.clear();

	static int idx = 0;
	_name = layer->_name + int_to_string(idx);
	idx ++;
	_inputName = layer->_inputName;

	inputAmount = layer->inputAmount;
	inputImageDim = layer->inputImageDim;
	number = layer->number;
	channels =  layer->channels;
	height = layer->height;
	width = layer->width;
	outputSize = layer->outputSize;
	ActivationMode = layer->ActivationMode;

	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&dstData, number * channels * height * width * sizeof(float));
	MemoryMonitor::instanceObject()->gpuMallocMemory((void**)&diffData, number * channels * height * width * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(dstData, layer->dstData, number * channels * width * sizeof(float));
	MemoryMonitor::instanceObject()->gpu2gpu(diffData, layer->diffData, number * channels * height * width * sizeof(float));

	cout<<"act deep copy"<<endl;
	this->createHandles();
}

void activationLayer::forwardPropagation(string train_or_test)
{
	srcData = prevLayer[0]->dstData;

    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
		                                 cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
		                                 cuDNN_netWork<float>::instanceObject()->GetDataType(),
		                                 number,
		                                 channels,
		                                 height,
		                                 width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	MemoryMonitor::instanceObject()->gpuMemoryMemset(dstData,number*channels*height*width*sizeof(float));
	float alpha = 1.0f;
	float beta = 0.0f;
	checkCUDNN(hipdnnActivationForward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                          ActivationMode,
			                          &alpha,
			                          srcTensorDesc,
			                          srcData,
			                          &beta,
			                          dstTensorDesc,
			                          dstData));
}


void activationLayer::backwardPropagation(float Momentum)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(dstDiffTensorDesc,
				                          cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
				                          cuDNN_netWork<float>::instanceObject()->GetDataType(),
				                          number,
				                          channels,
				                          height,
				                          width));

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
			                              cuDNN_netWork<float>::instanceObject()->GetTensorFormat(),
			                              cuDNN_netWork<float>::instanceObject()->GetDataType(),
			                              number,
			                              channels,
			                              height,
			                              width));

	float alpha = 1.0f;
	float beta = 0.0f;
	int nIndex = m_nCurBranchIndex;
	checkCUDNN(hipdnnActivationBackward(cuDNN_netWork<float>::instanceObject()->GetcudnnHandle(),
			                           ActivationMode,
			                           &alpha,
			                           dstTensorDesc,
			                           dstData,
			                           srcDiffTensorDesc,
			                           nextLayer[nIndex]->diffData,
			                           srcTensorDesc,
			                           srcData,
			                           &beta,
			                           dstDiffTensorDesc,
			                           diffData));

}

void activationLayer::destroyHandles()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(srcDiffTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(dstDiffTensorDesc));

}
